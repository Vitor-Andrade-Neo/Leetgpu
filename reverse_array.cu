#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void reverse_array(float* input, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N/2){
        float aux = input[N - id - 1];
        input[N - id - 1] = input[id];
        input[id] = aux;
    }
}

// input is device pointer
void solve(float* input, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    reverse_array<<<blocksPerGrid, threadsPerBlock>>>(input, N);
    hipDeviceSynchronize();
}
